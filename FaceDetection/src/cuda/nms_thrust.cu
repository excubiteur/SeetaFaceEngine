#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include<algorithm>

#include "common.h"

struct not_selected
{
	__device__
		bool operator()(int32_t x)
	{
		return x <= -1;
	}
};

struct BoxInfo
{
	seeta::Rect bbox;
	double score;
	int32_t original_index;
};

void CudaNonMaximumSuppression(const std::vector<seeta::FaceInfo>&data, std::vector<seeta::FaceInfo>* bboxes_nms, float iou_thresh)
{
	auto size = data.size();
	thrust::host_vector<BoxInfo> boxes(size);
	for (int i = 0; i < size; ++i)
	{
		BoxInfo box;
		box.bbox = data[i].bbox;
		box.score = data[i].score;
		box.original_index = i;
		boxes.push_back(box);
	}
	thrust::device_vector<BoxInfo> faces = boxes;
	BoxInfo * faces_ptr = thrust::raw_pointer_cast(faces.data());
	thrust::device_vector<int32_t> selected_indices(data.size(), -1);
	thrust::device_vector<double> scores(data.size(), 0.0);
	thrust::device_vector<int32_t> device_original_indices(data.size());

	thrust::sort(faces.begin(), faces.end(),[] __device__(const BoxInfo & a, const BoxInfo & b) {
		return a.score > b.score;
	});

	int32_t select_idx = 0;
	while (select_idx < size)
	{
		selected_indices[select_idx] = select_idx;
		int32_t start_idx = select_idx + 1;
		thrust::transform(
			thrust::make_zip_iterator(thrust::make_tuple(faces.begin() + start_idx, selected_indices.begin() + start_idx, scores.begin() + start_idx)),
			thrust::make_zip_iterator(thrust::make_tuple(faces.end(), selected_indices.end(), scores.end())),
			thrust::make_zip_iterator(thrust::make_tuple(selected_indices.begin() + start_idx, scores.begin() + start_idx)),
			[select_idx, iou_thresh, faces_ptr] __device__(const thrust::tuple<BoxInfo, int32_t, double>&input) {
			seeta::Rect select_bbox = faces_ptr[select_idx].bbox;
			float area1 = static_cast<float>(select_bbox.width * select_bbox.height);
			float x1 = static_cast<float>(select_bbox.x);
			float y1 = static_cast<float>(select_bbox.y);
			float x2 = static_cast<float>(select_bbox.x + select_bbox.width - 1);
			float y2 = static_cast<float>(select_bbox.y + select_bbox.height - 1);

			if(input.get<1>() >= 0)
				return thrust::make_tuple(input.get<1>(), input.get<2>());
			else
			{
				const seeta::Rect & bbox_i = input.get<0>().bbox;
				
				float x = x1 < static_cast<float>(bbox_i.x) ? static_cast<float>(bbox_i.x):x1;
				float y = y1 < static_cast<float>(bbox_i.y) ? static_cast<float>(bbox_i.y) : y1;
				auto temp_w = static_cast<float>(bbox_i.x + bbox_i.width - 1);
				auto temp_h = static_cast<float>(bbox_i.y + bbox_i.height - 1);
				float w = (x2 < temp_w ? x2 : temp_w) - x + 1;
				float h = (y2 < temp_h ? y2 : temp_h) - y + 1;

				if (w <= 0 || h <= 0)
					return thrust::make_tuple(input.get<1>(), input.get<2>());
				else
				{
					float area2 = static_cast<float>(bbox_i.width * bbox_i.height);
					float area_intersect = w * h;
					float area_union = area1 + area2 - area_intersect;
					if (static_cast<float>(area_intersect) / area_union > iou_thresh) {
						return thrust::make_tuple(select_idx, input.get<0>().score);
					}
					else {
						return thrust::make_tuple(input.get<1>(), input.get<2>());
					}
				}
			}
		});
		auto next = thrust::find_if(selected_indices.begin() + start_idx, selected_indices.end(), not_selected());
		select_idx = next - selected_indices.begin();
	}
	thrust::sort(
		thrust::make_zip_iterator(thrust::make_tuple(selected_indices.begin(), scores.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(selected_indices.end(), scores.end())),
		[] __device__(const thrust::tuple<int32_t, double>&a, const thrust::tuple<int32_t, double>&b) { return a.get<0>() < b.get<0>();  });

	thrust::device_vector<int32_t> device_final_indices(size);
	thrust::device_vector<double> device_final_scores(size);
	auto reduced_size_pair = thrust::reduce_by_key(selected_indices.begin(), selected_indices.end(), scores.begin(), device_final_indices.begin(), device_final_scores.begin());
	int reduced_size = reduced_size_pair.first - device_final_indices.begin();
	thrust::host_vector<int32_t> final_indices = device_final_indices;
	thrust::host_vector<double> final_scores = device_final_scores;
	thrust::transform(faces.begin(), faces.end(), device_original_indices.begin(), [] __device__(const BoxInfo&box) { return box.original_index; });
	thrust::host_vector<int32_t> original_indices = device_original_indices;
	for(int i = 0 ; i < reduced_size; ++i)
	{ 
		int index = final_indices[i];
		bboxes_nms->push_back(data[original_indices[index]]);
		bboxes_nms->back().score += final_scores[i];
	}
}